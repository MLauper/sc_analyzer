#include "hip/hip_runtime.h"
#include "lib_image_acquisition.h"
#include <cstdio>

/*! \brief Dummy CUDA kernel with no practical use
 *
 * This kernel does not run anything on the GPU, but can be used
 * to test if it is possible to properly lunch a kernel on the GPU.
 */
//__global__ void dummyKernel()
//{
//	return;
//}

void printSomething()
{
	printf("something");
}

__global__ void dummyKernel()
{
	return;
}