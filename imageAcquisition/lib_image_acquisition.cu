#include "hip/hip_runtime.h"
#include "lib_image_acquisition.h"

/*! \brief Dummy CUDA kernel with no practical use
 *
 * This kernel does not run anything on the GPU, but can be used
 * to test if it is possible to properly lunch a kernel on the GPU.
 */
//__global__ void dummyKernel()
//{
//	return;
//}
